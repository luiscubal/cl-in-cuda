#include "hip/hip_runtime.h"
#include "cl_device_assist.cuh"
#include "cl_interface_shared.h"

__global__ void clcuda_func_for_loops(int32_t *var_A, int32_t var_N, CommonKernelData data)
{
	if (blockIdx.x * blockDim.x + threadIdx.x >= data.totalX) return;
	if (blockIdx.y * blockDim.y + threadIdx.y >= data.totalY) return;
	if (blockIdx.z * blockDim.z + threadIdx.z >= data.totalZ) return;
	
	for (int32_t var_i = 0; var_i < var_N; var_i++)
	{
		var_A[(var_N * clcuda_builtin_get_global_size(0, data)) + clcuda_builtin_get_global_id(0, data)] = 1;
	}
}

KERNEL_LAUNCHER void clcuda_launcher_for_loops(struct _cl_kernel *desc)
{
	dim3 num_grids = dim3(desc->gridX, desc->gridY, desc->gridZ);
	dim3 local_size = dim3(desc->localX, desc->localY, desc->localZ);
	
	clcuda_func_for_loops<<<num_grids, local_size>>>(
		(int32_t*) desc->arg_data[0],
		*(int32_t*) desc->arg_data[1],
		CommonKernelData(desc->totalX, desc->totalY, desc->totalZ)
	);
}

