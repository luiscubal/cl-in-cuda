#include "hip/hip_runtime.h"
#include "cl_device_assist.cuh"
#include "cl_interface_shared.h"

__global__ void clcuda_func_for_loops(int *var_A, int var_N, CommonKernelData data)
{
	if (blockIdx.x * blockDim.x + threadIdx.x >= data.totalX) return;
	if (blockIdx.y * blockDim.y + threadIdx.y >= data.totalY) return;
	if (blockIdx.z * blockDim.z + threadIdx.z >= data.totalZ) return;

	for (int var_i = 0; var_i < var_N; var_i++)
	{
		var_A[(var_N * clcuda_builtin_get_global_size(0, data)) + clcuda_builtin_get_global_id(0, data)] = 1;
	}
}

KERNEL_LAUNCHER void clcuda_launcher_for_loops(struct _cl_kernel *desc)
{
	dim3 num_grids = dim3(desc->gridX, desc->gridY, desc->gridZ);
	dim3 local_size = dim3(desc->localX, desc->localY, desc->localZ);

	clcuda_func_for_loops<<<num_grids, local_size>>>(
		(int*) desc->arg_data[0],
		*(int*) desc->arg_data[1],
		CommonThreadData(desc->totalX, desc->totalY, desc->totalZ)
	);
}

