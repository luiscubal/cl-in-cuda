#include "hip/hip_runtime.h"
#include "cl_device_assist.cuh"
#include "cl_interface_shared.h"

__global__ void clcuda_func_branches(int32_t *var_A, int32_t *var_B, CommonKernelData data)
{
	if (blockIdx.x * blockDim.x + threadIdx.x >= data.totalX) return;
	if (blockIdx.y * blockDim.y + threadIdx.y >= data.totalY) return;
	if (blockIdx.z * blockDim.z + threadIdx.z >= data.totalZ) return;
	
	size_t var_i = clcuda_builtin_get_global_id(0, data);
	if (var_A[var_i] > 0)
	{
		var_B[var_i] = var_A[var_i];
	} else
	{
		var_B[var_i] = (-var_A[var_i]);
	}
}

KERNEL_LAUNCHER void clcuda_launcher_branches(struct _cl_kernel *desc)
{
	dim3 num_grids = dim3(desc->gridX, desc->gridY, desc->gridZ);
	dim3 local_size = dim3(desc->localX, desc->localY, desc->localZ);
	
	clcuda_func_branches<<<num_grids, local_size>>>(
		(int32_t*) desc->arg_data[0],
		(int32_t*) desc->arg_data[1],
		CommonThreadData(desc->totalX, desc->totalY, desc->totalZ)
	);
}

