#include "hip/hip_runtime.h"
#include "cl_device_assist.cuh"
#include "cl_interface_shared.h"

/*
Some comment
*/
// Single line
__global__ void clcuda_func_comment(int32_t *var_A, int32_t *var_B, int32_t *var_C, CommonKernelData data)
{
	if (blockIdx.x * blockDim.x + threadIdx.x >= data.totalX) return;
	if (blockIdx.y * blockDim.y + threadIdx.y >= data.totalY) return;
	if (blockIdx.z * blockDim.z + threadIdx.z >= data.totalZ) return;
	
	size_t var_i = clcuda_builtin_get_global_id(0, data);
	
	
	var_C[var_i] = (var_A[var_i] + var_B[var_i]);
}

KERNEL_LAUNCHER void clcuda_launcher_comment(struct _cl_kernel *desc)
{
	dim3 num_grids = dim3(desc->gridX, desc->gridY, desc->gridZ);
	dim3 local_size = dim3(desc->localX, desc->localY, desc->localZ);
	
	clcuda_func_comment<<<num_grids, local_size>>>(
		(int32_t*) desc->arg_data[0],
		(int32_t*) desc->arg_data[1],
		(int32_t*) desc->arg_data[2],
		CommonKernelData(desc->totalX, desc->totalY, desc->totalZ)
	);
}

