#include "hip/hip_runtime.h"
#include "cl_device_assist.cuh"
#include "cl_interface_shared.h"

__global__ void clcuda_func_while_loop(int *var_A, int var_a, CommonKernelData data)
{
	if (blockIdx.x * blockDim.x + threadIdx.x >= data.totalX) return;
	if (blockIdx.y * blockDim.y + threadIdx.y >= data.totalY) return;
	if (blockIdx.z * blockDim.z + threadIdx.z >= data.totalZ) return;

	size_t var_i = get_global_id(0, data);
	while (var_A[var_i] < var_a)
	{
		var_A[var_i] *= 2;
	}
}

KERNEL_LAUNCHER void clcuda_launcher_while_loop(struct _cl_kernel *desc)
{
	dim3 num_grids = dim3(desc->gridX, desc->gridY, desc->gridZ);
	dim3 local_size = dim3(desc->localX, desc->localY, desc->localZ);

	clcuda_func_while_loop<<<num_grids, local_size>>>(
		(int*) desc->arg_data[0],
		*(int*) desc->arg_data[1],
		CommonThreadData(desc->totalX, desc->totalY, desc->totalZ)
	);
}

