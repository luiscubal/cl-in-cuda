#include "hip/hip_runtime.h"
#include "cl_device_assist.cuh"
#include "cl_interface_shared.h"

__global__ void clcuda_func_while_loop(int32_t *var_A, int32_t var_a, CommonKernelData data)
{
	if (blockIdx.x * blockDim.x + threadIdx.x >= data.totalX) return;
	if (blockIdx.y * blockDim.y + threadIdx.y >= data.totalY) return;
	if (blockIdx.z * blockDim.z + threadIdx.z >= data.totalZ) return;
	
	size_t var_i = clcuda_builtin_get_global_id(0, data);
	while (var_A[var_i] < var_a)
	{
		var_A[var_i] *= 2;
	}
}

KERNEL_LAUNCHER void clcuda_launcher_while_loop(struct _cl_kernel *desc, float *elapsedMs)
{
	dim3 num_grids = dim3(desc->gridX, desc->gridY, desc->gridZ);
	dim3 local_size = dim3(desc->localX, desc->localY, desc->localZ);
	
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	
	hipEventRecord(start);
	clcuda_func_while_loop<<<num_grids, local_size>>>(
		(int32_t*) desc->arg_data[0],
		*(int32_t*) desc->arg_data[1],
		CommonKernelData(desc->totalX, desc->totalY, desc->totalZ)
	);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(elapsedMs, start, end);
}

