#include "hip/hip_runtime.h"
#include "cl_device_assist.cuh"
#include "cl_interface_shared.h"

typedef size_t clcuda_type_mytype;

__global__ void clcuda_func_type(int32_t *var_A, float *var_B, int32_t *var_C, CommonKernelData data)
{
	if (blockIdx.x * blockDim.x + threadIdx.x >= data.totalX) return;
	if (blockIdx.y * blockDim.y + threadIdx.y >= data.totalY) return;
	if (blockIdx.z * blockDim.z + threadIdx.z >= data.totalZ) return;
	
	clcuda_type_mytype var_i = clcuda_builtin_get_global_id(0, data);
	var_C[var_i] = (var_A[var_i] + ((int32_t) var_B[var_i]));
}

KERNEL_LAUNCHER void clcuda_launcher_type(struct _cl_kernel *desc, float *elapsedMs)
{
	dim3 num_grids = dim3(desc->gridX, desc->gridY, desc->gridZ);
	dim3 local_size = dim3(desc->localX, desc->localY, desc->localZ);
	
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	
	hipEventRecord(start);
	clcuda_func_type<<<num_grids, local_size>>>(
		(int32_t*) desc->arg_data[0],
		(float*) desc->arg_data[1],
		(int32_t*) desc->arg_data[2],
		CommonKernelData(desc->totalX, desc->totalY, desc->totalZ)
	);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(elapsedMs, start, end);
}

