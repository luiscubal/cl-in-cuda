#include "hip/hip_runtime.h"
#include "cl_device_assist.cuh"
#include "cl_interface_shared.h"

__global__ void clcuda_func_manualbound(float *var_A, int var_N, CommonKernelData data)
{
	if (blockIdx.x * blockDim.x + threadIdx.x >= data.totalX) return;
	if (blockIdx.y * blockDim.y + threadIdx.y >= data.totalY) return;
	if (blockIdx.z * blockDim.z + threadIdx.z >= data.totalZ) return;
	
	size_t var_global_id;
	var_global_id = clcuda_builtin_get_global_id(0U, data);
	if (var_global_id < var_N)
	{
		var_A[var_global_id] = 1.0;
	}
}

KERNEL_LAUNCHER void clcuda_launcher_manualbound(struct _cl_kernel *desc)
{
	dim3 num_grids = dim3(desc->gridX, desc->gridY, desc->gridZ);
	dim3 local_size = dim3(desc->localX, desc->localY, desc->localZ);
	
	clcuda_func_manualbound<<<num_grids, local_size>>>(
		(float*) desc->arg_data[0],
		*(int*) desc->arg_data[1],
		CommonThreadData(desc->totalX, desc->totalY, desc->totalZ)
	);
}

